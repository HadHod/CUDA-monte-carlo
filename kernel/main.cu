
#include <hip/hip_runtime.h>
__global__
void estimatePI(int* rslt_data, const double* rnd_values1, const double* rnd_values2) {
    const unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    const double v1 = rnd_values1[tid];
    const double v2 = rnd_values2[tid];

    rslt_data[tid] = (v1 * v1 + v2 * v2 <= 1) ? 1 : 0;
}