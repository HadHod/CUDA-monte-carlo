
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

// __global__ void setup_kernel ( curandState * state, unsigned long seed ) {
//     int id = threadIdx.x;
//     curand_init ( seed, id, 0, &state[id] );
// }

// __global__ void generate( curandState* globalState ) {
//     int ind = threadIdx.x;
//     curandState localState = globalState[ind];
//     float RANDOM = curand_uniform( &localState );
//     globalState[ind] = localState;
// }

__global__
void estimatePI(double* resultData) {
    const unsigned int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

    hiprandState state;
    // curand_init(tid, tid, 0, &state);
    float RANDOM = hiprand_uniform( &state );

    printf("Seed: %f %i\n", RANDOM);

    resultData[tid] = true ? 1 : 0;
}