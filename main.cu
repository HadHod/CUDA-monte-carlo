#include "hip/hip_runtime.h"
#include <iostream>

#include "kernel/main.cu"
#include "numberGenerator/NumberGenerator.h"

using namespace std;

int main(int argc, char* argv[]) {
    const int size = 10;

    // TODO calculate max threads on graphic card

    NumberGenerator ng;
    // TODO random values <-1, 1>
    const double* randomValues1 = ng.getRandomNumbers(size);
    const double* randomValues2 = ng.getRandomNumbers(size);
    double* resultValues;

    double* dev_randomValues1;
    double* dev_randomValues2;
    double* dev_resultValues;

    const size_t MEMORY_SIZE = size * sizeof(double);
    hipMalloc((void**) &dev_randomValues1, MEMORY_SIZE);
    hipMalloc((void**) &dev_randomValues2, MEMORY_SIZE);
    hipMalloc((void**) &dev_resultValues, MEMORY_SIZE);

    hipMemcpy(dev_randomValues1, randomValues1, MEMORY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_randomValues2, randomValues2, MEMORY_SIZE, hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventRecord(start, 0);

    estimatePI<<<10, 10>>>();
    hipDeviceSynchronize();

    hipEventCreate(&stop);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    cout << "Elapsed time: " << elapsedTime << " ms\n";

    hipMemcpy(resultValues, dev_resultValues, MEMORY_SIZE, hipMemcpyDeviceToHost);

    // TODO change to functional
    int numOfPointsInCircle = 0;
    for (int i=0; i<size; i++) {
        numOfPointsInCircle += resultValues[i];
    }

    const double result = 4.0 * (double) numOfPointsInCircle / (double) size;
    cout << "estimated value of PI: " << result << endl;

    delete[] randomValues1;
    delete[] randomValues2;
    delete[] resultValues;

    hipFree(dev_randomValues1);
    hipFree(dev_randomValues2);
    hipFree(dev_resultValues);

    return EXIT_SUCCESS;
}