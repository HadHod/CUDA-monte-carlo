#include <iostream>

#include "kernel/main.cu"

using namespace std;

int main(int argc, char* argv[]) {

    NumberGenerator ng;
    const double* randomValues1 = ng.getRandomNumbers(ARRAY_SIZE);
    const double* randomValues2 = ng.getRandomNumbers(ARRAY_SIZE);
    double* resultValues;

    double* dev_randomValues1;
    double* dev_randomValues2;
    double* dev_resultValues;

    const int size = 10;
    const size_t MEMORY_SIZE = size * sizeof(double);
    hipMalloc((void**) &dev_randomValues1, MEMORY_SIZE);
    hipMalloc((void**) &dev_randomValues2, MEMORY_SIZE);
    hipMalloc((void**) &dev_resultValues, MEMORY_SIZE);

    hipMemcpy(dev_a, a, MEMORY_SIZE, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, MEMORY_SIZE, hipMemcpyHostToDevice);

    estimatePI<<<10, 10>>>();

    cout << "PI: " << "" << endl;

    return EXIT_SUCCESS;
}