#include <iostream>

#include "kernel/main.cu"

using namespace std;

int main(int argc, char* argv[]) {
    const int size = 10;

    double* resultValues = new double[size];
    double* dev_resultValues;

    const size_t MEMORY_SIZE = size * sizeof(double);
    hipMalloc((void**) &dev_resultValues, MEMORY_SIZE);

    estimatePI<<<1, 10>>>(dev_resultValues);
    hipDeviceSynchronize();

    hipMemcpy(resultValues, dev_resultValues, MEMORY_SIZE, hipMemcpyDeviceToHost);

    // TODO change to functional
    int numOfPointsInCircle = 0;
    for (int i=0; i<size; i++) {
        numOfPointsInCircle += resultValues[i];
    }

    const double result = 4.0 * (double) numOfPointsInCircle / (double) size;
    cout << "estimated value of PI: " << result << endl;

    delete[] resultValues;

    hipFree(dev_resultValues);

    return EXIT_SUCCESS;
}